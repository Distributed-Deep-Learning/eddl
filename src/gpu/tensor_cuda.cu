// This file is part of EDDLL an European Distributed Deep Learning Library.
// Developed within the DeepHealth project.
// Boosting AI in Europe.
//
// The MIT License (MIT)
//
// Copyright (c) 2019 Roberto Paredes Palacios, <rparedes@dsic.upv.es>

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <stdio.h>
#include "tensor_cuda.h"

hipblasHandle_t hcublas[64];
hiprandGenerator_t random_generator[64];
hipblasStatus_t bstatus;
hiprandStatus_t rstatus;



void check_cuda(hipError_t err,char *msg)
{
  if(err!=hipSuccess)
  {
     fprintf(stderr,"Cuda Error: %s\n",msg);
     exit(0);
  }

}

void gpu_init(int device)
{

  int nDevices;
  hipGetDeviceCount(&nDevices);

  if (device>nDevices)
   {
    fprintf(stderr,"Error. GPU %d not available. Number of available GPU is %d. Further information running nvidia-smi\n",device,nDevices);
    exit(-1);
   }

  fprintf(stderr,"Selecting GPU device %d\n",device);
  hipSetDevice(device);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop,device);

  fprintf(stderr,"EDDLL is running on GPU device %d, %s\n",device,prop.name);


  /// CUBLAS
  bstatus=hipblasCreate(&(hcublas[device]));
  // try to init cublas several times
  int i=0;
  while ((bstatus!=  HIPBLAS_STATUS_SUCCESS)&&(i<10)) {
    bstatus=hipblasCreate(&(hcublas[device]));
    i++;
    fprintf(stderr,".");
  }
  fprintf(stderr,"\n");

  if ( bstatus!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas Create\n");
     exit(1);

  }
  fprintf(stderr,"CuBlas initialized on GPU device %d, %s\n",device,prop.name);

  bstatus = hipblasSetAtomicsMode(hcublas[device],HIPBLAS_ATOMICS_NOT_ALLOWED);
  if ( bstatus!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Problem in cuBlas execution getting: NOT IMPLEMENTED \n");
     exit(1);

  }

  // CURAND
  rstatus=hiprandCreateGenerator(&(random_generator[device]),HIPRAND_RNG_PSEUDO_MRG32K3A);
  if (rstatus != HIPRAND_STATUS_SUCCESS)
	{
    fprintf(stderr,"Error creating random numbers on gpu\n");
    exit(-1);
   }
  rstatus=hiprandSetPseudoRandomGeneratorSeed(random_generator[device],1234);

  if (rstatus != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr,"Error seeting the seed for program\n");
      exit(-1);
  }
  fprintf(stderr,"CuRand initialized on GPU device %d, %s\n",device,prop.name);



}

void gpu_set_device(int device)
{
   hipSetDevice(device);
}


///////////////

float* gpu_create_tensor(int size)
{
  float* devicePointer;
  check_cuda(hipMalloc((void**)&devicePointer,size*sizeof(float)),"create_tensor");
  return devicePointer;
}
void gpu_delete_tensor(float* p)
{
  check_cuda(hipFree(p),"delete_tensor");
}
