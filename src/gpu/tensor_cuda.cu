#include <stdio.h>
#include "tensor_cuda.h"


void check_cuda(hipError_t err,char *msg)
{
  if(err!=hipSuccess)
  {
     fprintf(stderr,"Cuda Error: %s\n",msg);
     exit(0);
  }

}

float* create_tensor(int size)
{
  float* devicePointer;
  check_cuda(hipMalloc((void**)&devicePointer,size*sizeof(float)),"create_tensor");
  return devicePointer;
}


void delete_tensor(float* p)
{
  check_cuda(hipFree(p),"delete_tensor");
}
