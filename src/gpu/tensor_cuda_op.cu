#include "hip/hip_runtime.h"
// This file is part of EDDLL an European Distributed Deep Learning Library.
// Developed within the DeepHealth project.
// Boosting AI in Europe.
//
// The MIT License (MIT)
//
// Copyright (c) 2019 Roberto Paredes Palacios, <rparedes@dsic.upv.es>

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "../tensor.h"
#include "tensor_cuda.h"
#include "tensor_kernels.h"

extern hipblasHandle_t hcublas[64];
extern hiprandGenerator_t random_generator[64];

static const char *_curandGetErrorEnum(hiprandStatus_t error)
{
    switch (error)
    {
        case HIPRAND_STATUS_ALLOCATION_FAILED:
            return "HIPRAND_STATUS_ALLOCATION_FAILED";
            break;
        case HIPRAND_STATUS_INITIALIZATION_FAILED:
            return "HIPRAND_STATUS_INITIALIZATION_FAILED";
            break;
        case HIPRAND_STATUS_VERSION_MISMATCH:
            return "HIPRAND_STATUS_VERSION_MISMATCH";
            break;

        case HIPRAND_STATUS_TYPE_ERROR:
            return "HIPRAND_STATUS_TYPE_ERROR";
            break;

        case HIPRAND_STATUS_OUT_OF_RANGE:
            return "HIPRAND_STATUS_OUT_OF_RANGE";
            break;

        case HIPRAND_STATUS_PREEXISTING_FAILURE:
            return "HIPRAND_STATUS_PREEXISTING_FAILURE";
            break;

        case HIPRAND_STATUS_NOT_INITIALIZED:
            return "HIPRAND_STATUS_NOT_INITIALIZED";
            break;

        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
            return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
            break;
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
            return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        default:
	    fprintf(stderr,"Not all hiprand errors here %d\n",error);
	    exit(-1);
    }

}


void check_cublas(hipblasStatus_t status, const char *f)
{
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Error in cublas execution in %s\n",f);
     exit(1);
  }
}

void check_curand(hiprandStatus_t status, const char *f)
{
  if ( status!=  HIPRAND_STATUS_SUCCESS)
  {
     fprintf(stderr,"Error in hiprand execution in %s\n",_curandGetErrorEnum(status));
     exit(1);
  }
}


///////////////////////////////////////////
void gpu_set(Tensor *A,float v) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  set<<<dimGrid,dimBlock>>>(A->ptr,v,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"set");

}

///////////////////////////////////////////
void gpu_mult(Tensor *A,float v) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  mult<<<dimGrid,dimBlock>>>(A->ptr,v,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"mult");

}
///////////////////////////////////////////
void gpu_sum(Tensor *A,float v) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  sum<<<dimGrid,dimBlock>>>(A->ptr,v,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"sum");

}

///////////////////////////////////////////
void gpu_log(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  log<<<dimGrid,dimBlock>>>(A->ptr,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"log");

}

///////////////////////////////////////////
void gpu_exp(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  exp<<<dimGrid,dimBlock>>>(A->ptr,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"exp");

}

///////////////////////////////////////////
void gpu_sqrt(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  sqrt<<<dimGrid,dimBlock>>>(A->ptr,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"sqrt");

}

///////////////////////////////////////////
void gpu_sqr(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  sqr<<<dimGrid,dimBlock>>>(A->ptr,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"sqr");

}


///////////////////////////////////////////
void gpu_mask(Tensor *A,float v) {

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  mask<<<dimGrid,dimBlock>>>(A->ptr,v,A->sizes[0],c);
  check_cuda(hipDeviceSynchronize(),"mask");

}

void gpu_total_sum(Tensor *A,float *tot)
{
  float *total;
  int device=A->gpu_device;
  hipSetDevice(device);

  int r=A->sizes[0];
  int c=A->tam/r;

  dim3 dimBlock(r);
  dim3 dimGrid(1);
  long int ops = r;


  check_cuda(hipMalloc((void**)&total,sizeof(float)),"create float in total_sum");
  reduce_array_sum<<<dimGrid,dimBlock,ops*sizeof(float)>>>(A->ptr,ops,c,total);
  check_cuda(hipMemcpy(tot,total,sizeof(float),hipMemcpyDeviceToHost),"error copy in total_sum");

  check_cuda(hipFree(total),"delete float in total_sum");
}

///////////////////////////////////////////
void gpu_copy_to_gpu(float *nptr,Tensor *A)
{
  int device=A->gpu_device;
  hipSetDevice(device);
  check_cuda(hipMemcpy(A->ptr,nptr,A->tam*sizeof(float),hipMemcpyHostToDevice),"gpu_copy_to_gpu");
}

///////////////////////////////////////////
void gpu_copy_from_gpu(Tensor *A,float *nptr)
{
  int device=A->gpu_device;
  hipSetDevice(device);
  check_cuda(hipMemcpy(nptr,A->ptr,A->tam*sizeof(float),hipMemcpyDeviceToHost),"gpu_copy_to_gpu");
}

///////////////////////////////////////////
void gpu_copy_gpu(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);
  check_cuda(hipMemcpy(B->ptr,A->ptr,A->tam*sizeof(float),hipMemcpyDeviceToDevice),"gpu_copy_gpu");
}


///////////////////////////////////////////

void gpu_mult2D(Tensor *A, int tA, Tensor *B, int tB, Tensor *C,int incC)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  float alfa=1.0;
  float beta=(float)incC;

  hipblasOperation_t trA = HIPBLAS_OP_N;
  hipblasOperation_t trB = HIPBLAS_OP_N;

  int ldA=A->sizes[1];
  int ldB=B->sizes[1];
  int ldC=B->sizes[1];
  int m=B->sizes[1];
  int n=A->sizes[0];
  int k=B->sizes[0];


  if (tA)
  {
    trA = HIPBLAS_OP_T;
  	n=A->sizes[1];
  }
  if (tB)
    {
  	trB = HIPBLAS_OP_T;
    m=B->sizes[0];
  	k=B->sizes[1];
    ldC=B->sizes[0];
    }

  check_cublas(hipblasSgemm(hcublas[device],trB,trA,m,n,k,&alfa,B->ptr,ldB,A->ptr,ldA,&beta,C->ptr,ldC),"mult2D");

}


void gpu_el_mult(Tensor *A, Tensor *B, Tensor *C,int incC)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);


  el_mult<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,incC,A->sizes[0],c);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");
}

void gpu_el_div(Tensor *A, Tensor *B, Tensor *C,int incC)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);


  el_mult<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,incC,A->sizes[0],r);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");
}


///////////////////////////////////////////
void gpu_sum(float scA,Tensor *A, float scB,Tensor *B, Tensor *C,int incC)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  sum<<<dimGrid,dimBlock>>>(scA,A->ptr,scB,B->ptr,C->ptr,incC,A->tam);
  check_cuda(hipDeviceSynchronize(),"sum");
}
///////////////////////////////////////////
void gpu_sum2D(float scA,Tensor *A, float scB,Tensor *B, Tensor *C,int incC)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  int m=A->sizes[1];
  int n=B->sizes[0];
  int ldA=A->sizes[1];
  int ldB=B->sizes[1];
  int ldC=A->sizes[1];

  float alfa=scA;
  float beta=scB;
  float one=1.0;


  if (incC){
    check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,A->ptr,ldA,&one,C->ptr,ldB,C->ptr,ldC),"sum2D");
    check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,B->ptr,ldA,&one,C->ptr,ldB,C->ptr,ldC),"sum2D");
  }
  else
    check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,A->ptr,ldA,&beta,B->ptr,ldB,C->ptr,ldC),"sum2D");

}


///////////////////////////////////////////
void gpu_sum2D_rowwise(Tensor *A, Tensor *B, Tensor *C)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  dim3 dimGrid(A->sizes[0]);
  dim3 dimBlock(A->sizes[1]);


  sum_mat_row<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->sizes[0],A->sizes[1]);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");

}
///////////////////////////////////////////
void gpu_sum2D_colwise(Tensor *A, Tensor *B, Tensor *C)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  dim3 dimGrid(A->sizes[0]);
  dim3 dimBlock(A->sizes[1]);

  sum_mat_col<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->sizes[0],A->sizes[1]);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");

}


///////////////////////////////////////////

void gpu_reduce_sum2D(Tensor *A,Tensor *B,int axis,int incB)
{

  int device=A->gpu_device;
  hipSetDevice(device);

  dim3 dimGrid(A->sizes[0]);
  dim3 dimBlock(A->sizes[1]);

  if (!incB) gpu_set(B,0.0);

  reduce_sum2D<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,A->sizes[0],A->sizes[1],axis);


  check_cuda(hipDeviceSynchronize(),"reduce_sum2D");
}
///////////////////////////////////////////

///////////////////////////////////////////
////// RAND
///////////////////////////////////////////
void gpu_rand_uniform(Tensor *A, float v)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->tam),"gpu_rand_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

  gpu_mult(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

}
///////////////////////////////////////////
void gpu_rand_suniform(Tensor *A, float v)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->tam),"gpu_rand_suniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_suniform");

  gpu_mult(A,2*v);
  gpu_sum(A,-v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_suniform");

}

///////////////////////////////////////////
void gpu_rand_gaussian(Tensor *A, float m,float s)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  if (A->tam%2) {
    gpu_set(A,0.0);
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->tam-1,m,s),"gpu_rand_gaussian");
  }
  else
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->tam,m,s),"gpu_rand_gaussian");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_gaussian");

}

///////////////////////////////////////////
void gpu_rand_binary(Tensor *A, float v)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->tam),"gpu_rand_binary");

  gpu_mask(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_binary");

}



///////////////////////////////////////////
void gpu_cent(Tensor *A,Tensor *B,Tensor *C)
{

  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  cent<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->tam);
  check_cuda(hipDeviceSynchronize(),"gpu_cent");

}

////////////////////////////////////
void gpu_accuracy(Tensor *A,Tensor *B,int *acc)
{
  int device=A->gpu_device;
  hipSetDevice(device);
  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  float* max_row=gpu_create_tensor(device,r);

  int *a;
  check_cuda(hipMalloc((void**)&a,sizeof(int)),"error hipMalloc in accuracy");
  hipMemset(a, 0, sizeof(int));

  accuracy<<<dimBlock,dimGrid>>>(A->ptr,B->ptr,max_row,c,r,a);
  check_cuda(hipMemcpy(acc,a,sizeof(float),hipMemcpyDeviceToHost),"error copy in accuracy");

  hipFree(a);
  gpu_delete_tensor(device,max_row);

}



////////////////////////////////////
void gpu_relu(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=A->sizes[0];
  c=A->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  relu<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,A->tam);
  check_cuda(hipDeviceSynchronize(),"gpu_relu");
}


void gpu_d_relu(Tensor *D,Tensor *I,Tensor *PD)
{
  int device=D->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=D->sizes[0];
  c=D->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  d_relu<<<dimGrid,dimBlock>>>(D->ptr,I->ptr,PD->ptr,D->tam);
  check_cuda(hipDeviceSynchronize(),"gpu_relu");
}

////////////////////////////////////
void gpu_softmax(Tensor *A,Tensor *B)
{
  int device=A->gpu_device;
  hipSetDevice(device);


/*
dimBlock.x=sp->row;
 dimGrid.x=1;
 long int ops = sp->col*sp->row;
long int sample_dim=sp->col;

double alfa=1;
float* auxE=NULL;
  ops=sp->row;
          auxE = makeTensor(sp->col,sp->row);
          set_sc(auxE, 0.0, sp);
  	Softmax<<<dimBlock,dimGrid>>>(E,N,auxE,sample_dim,ops);
*/

  int r,c;

  r=A->sizes[0];
  c=A->sizes[1];

  dim3 dimGrid(1);
  dim3 dimBlock(r);

  float* aux=gpu_create_tensor(device,A->tam);
  softmax<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,aux,c,A->tam);
  check_cuda(hipDeviceSynchronize(),"gpu_relu");
  gpu_delete_tensor(device,aux);
}


void gpu_d_softmax(Tensor *D,Tensor *I,Tensor *PD)
{
  int device=D->gpu_device;
  hipSetDevice(device);

  int r,c;

  r=D->sizes[0];
  c=D->tam/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  d_relu<<<dimGrid,dimBlock>>>(D->ptr,I->ptr,PD->ptr,D->tam);
  check_cuda(hipDeviceSynchronize(),"gpu_relu");
}











////////////////////////////////////
