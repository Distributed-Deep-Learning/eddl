// This file is part of EDDLL an European Distributed Deep Learning Library.
// Developed within the DeepHealth project.
// Boosting AI in Europe.
//
// The MIT License (MIT)
//
// Copyright (c) 2019 Roberto Paredes Palacios, <rparedes@dsic.upv.es>

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "../tensor.h"
#include "tensor_cuda.h"
#include "tensor_kernels.h"

extern hipblasHandle_t hcublas[64];
extern hiprandGenerator_t random_generator[64];


void check_cublas(hipblasStatus_t status, char * func)
{
  if ( status!=  HIPBLAS_STATUS_SUCCESS)
  {
     fprintf(stderr,"Error in cublas execution in %s\n",func);
     exit(1);
  }
}


///////////////////////////////////////////

void gpu_mult2D(Tensor *A, int tA, Tensor *B, int tB, Tensor *C,int incC)
{
  int device=A->gpu_device;

  float alfa=1.0;
  float beta=(float)incC;

  hipblasOperation_t trA = HIPBLAS_OP_N;
  hipblasOperation_t trB = HIPBLAS_OP_N;

  int ldA=A->sizes[1];
  int ldB=B->sizes[1];
  int ldC=B->sizes[1];
  int m=B->sizes[1];
  int n=A->sizes[0];
  int k=B->sizes[0];


  if (tA)
  {
    trA = HIPBLAS_OP_T;
  	n=A->sizes[1];
  }
  if (tB)
    {
  	trB = HIPBLAS_OP_T;
    m=B->sizes[0];
  	k=B->sizes[1];
    ldC=B->sizes[0];
    }

  check_cublas(hipblasSgemm(hcublas[device],trB,trA,m,n,k,&alfa,B->gptr,ldB,A->gptr,ldA,&beta,C->gptr,ldC),"mult2D");

}

///////////////////////////////////////////
void gpu_sum2D_rowwise(Tensor *A, Tensor *B, Tensor *C)
{
  int device=A->gpu_device;
  hipSetDevice(device);

  dim3 dimGrid(A->sizes[1]);
  dim3 dimBlock(A->sizes[0]);


  sum_mat_row<<<dimBlock,dimGrid>>>(A->gptr,B->gptr,C->gptr,A->sizes[0],A->sizes[1]);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");

}

///////////////////////////////////////////
void gpu_sum2D(Tensor *A, Tensor *B, Tensor *C,int incC)
{
  int device=A->gpu_device;

  int m=A->sizes[1];
  int n=B->sizes[0];
  int ldA=A->sizes[1];
  int ldB=B->sizes[1];
  int ldC=A->sizes[1];

  float alfa=1.0;
  float beta=1.0;


  if (incC){
    check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,A->gptr,ldA,&beta,C->gptr,ldB,C->gptr,ldC),"sum2D");
    check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,B->gptr,ldA,&beta,C->gptr,ldB,C->gptr,ldC),"sum2D");
  }
  else
    check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,A->gptr,ldA,&beta,B->gptr,ldB,C->gptr,ldC),"sum2D");

}

///////////////////////////////////////////


///////////////////////////////////////////


///////////////////////////////////////////
