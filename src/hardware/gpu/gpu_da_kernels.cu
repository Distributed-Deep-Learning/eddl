#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.1
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"

__global__ void shift(float* A, float* B, int batch, int depth, int irows, int icols, int* shift, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        int Ai = Bi - shift[0];
        int Aj = Bj - shift[1];

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            if(mode==0){ // constant
                B[thread_id_x] = constant;
            }
        }
    }

}

__global__ void rotate(float* A, float* B, int batch, int depth, int irows, int icols, float angle, int* axis, bool reshape, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    // Not implemented
    if (thread_id_x < ops){
        B[thread_id_x] = constant;
    }
}

__global__ void scale(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* new_shape, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*orows*ocols;

    if (thread_id_x < ops){
        int offsets[2] = {0, 0};
        offsets[0] = (new_shape[0] - orows)/2.0f;
        offsets[1] = (new_shape[1] - ocols)/2.0f;

        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // Interpolate indices
        if(mode==2) { // Nearest
            int Ai = ((Bi + offsets[0]) * irows) / new_shape[0];
            int Aj = ((Bj + offsets[1]) * icols) / new_shape[1];

            int B_pos = b * B_stride[0] + c * B_stride[1] + Bi * B_stride[2] + Bj * B_stride[3];
            if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols) {
                int A_pos = b * A_stride[0] + c * A_stride[1] + Ai * A_stride[2] + Aj * A_stride[3];
                B[B_pos] = A[A_pos];
            } else {
                B[B_pos] = constant;  // Equivalent to constant
            }
        }

    }

}

__global__ void flip(float* A, float* B, int batch, int depth, int irows, int icols, int axis, bool apply){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);
        int B_pos = b*B_stride[0] + c*B_stride[1] + Bi*B_stride[2] + Bj*B_stride[3];

        if(apply){
            int pos[2] = {Bi, Bj}; pos[axis] = (irows-1) - pos[axis];
            int Ai = pos[0]; int Aj = pos[1];
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[B_pos] = A[A_pos];
        }else{
            B[B_pos] = A[B_pos];
        }

    }
}


__global__ void crop(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, int* offsets, float constant, bool inverse){
   long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
   long int ops = batch * depth*irows*icols;

   if (thread_id_x < ops){
       int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
       int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

       //--------------
       int b = thread_id_x / B_stride[0] % batch;
       int c = thread_id_x / B_stride[1] % depth;
       int Bi = thread_id_x / B_stride[2] % orows;
       int Bj = thread_id_x / B_stride[3] % ocols;

       // Compute coordinates
       int Ai = Bi + offsets[0];  // Start from the (0,0) of the cropping area
       int Aj = Bj + offsets[1];

       bool inRegion = Ai >= coords_from[0] && Ai <= coords_to[0] && Aj >= coords_from[1] && Aj <= coords_to[1];
       int B_pos = b*B_stride[0] + c*B_stride[1] + Bi*B_stride[2] + Bj*B_stride[3];  // We always walk through the whole B tensor

       if ((inRegion && !inverse) || (!inRegion && inverse)){
           int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
           B[B_pos] = A[A_pos];
       }else{
           B[B_pos] = constant;
       }

       
   }
}


__global__ void crop_scale(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_wc = coords_to[0]-coords_from[0]+1;
        int A_hc = coords_to[0]-coords_from[1]+1;

        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;

        // Interpolate indices
        if(mode==2) { // Nearest
            int Ai = (Bi * A_hc) / orows + coords_from[0];
            int Aj = (Bj * A_wc) / ocols + coords_from[1];

            int A_pos = b * A_stride[0] + c * A_stride[1] + Ai * A_stride[2] + Aj * A_stride[3];
            int B_pos = b * B_stride[0] + c * B_stride[1] + Bi * B_stride[2] + Bj * B_stride[3];

            B[B_pos] = A[A_pos];
        }
    }
}


__global__ void shift_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor_x, float* factor_y, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        int shift_x = (int)(icols * ((factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]));
        int shift_y = (int)(irows * ((factor_y[1]-factor_y[0]) * rnd[b+1] + factor_y[0]));

        int Ai = Bi - shift_y;
        int Aj = Bj - shift_x;

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            if(mode==0){ // constant
                B[thread_id_x] = constant;
            }
        }
    }

}

__global__ void rotate_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor, int* axis, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    // TODO: Implement
    if (thread_id_x < ops){
        B[thread_id_x] = constant;
    }
}

__global__ void scale_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*orows*ocols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        float scale = (factor[1]-factor[0]) * rnd[b] + factor[0];
        int new_shape_x = (int)(icols * scale);
        int new_shape_y = (int)(irows * scale);

        // Center crop (if the if the crop is smaller than B)
        int offsets[2] = {0, 0};
        offsets[0] = (new_shape_y - orows)/2.0f;
        offsets[1] = (new_shape_x - ocols)/2.0f;

        // Interpolate indices
        if(mode==2) { // Nearest
            int Ai = ((Bi + offsets[0]) * irows) / new_shape_y;
            int Aj = ((Bj + offsets[1]) * icols) / new_shape_x;

            int B_pos = b * B_stride[0] + c * B_stride[1] + Bi * B_stride[2] + Bj * B_stride[3];
            if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols) {
                int A_pos = b * A_stride[0] + c * A_stride[1] + Ai * A_stride[2] + Aj * A_stride[3];
                B[B_pos] = A[A_pos];
            } else {
                B[B_pos] = constant;  // Equivalent to constant
            }
        }
    }

}

__global__ void flip_random(float* A, float* B, int batch, int depth, int irows, int icols, int axis, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        bool apply = rnd[b] >= 0.5f;
        int B_pos = b*B_stride[0] + c*B_stride[1] + Bi*B_stride[2] + Bj*B_stride[3];

        if(apply){
            int pos[2] = {Bi, Bj}; pos[axis] = (irows-1) - pos[axis];
            int Ai = pos[0]; int Aj = pos[1];
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[B_pos] = A[A_pos];
        }else{
            B[B_pos] = A[B_pos];
        }

    }
}


__global__ void crop_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;
        // printf("A={%d, %d, %d, %d}\n", b, c, Ai, Aj);
        // printf("B={%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // Compute random coordinates
        int w = icols;
        int h = ocols;
        int x = (int)((icols-w) * rnd[b]);
        int y = (int)((irows-h) * rnd[b+1]);

        int coords_from_x = x;
        //int coords_to_x = x+w;
        int coords_from_y = y;
        //int coords_to_y = y+h;

        // Compute coordinates
        int Ai = Bi + coords_from_y;  // Start from the (0,0) of the cropping area
        int Aj = Bj + coords_from_x;

        int B_pos = b*B_stride[0] + c*B_stride[1] + Bi*B_stride[2] + Bj*B_stride[3];  // We always walk through the whole B tensor
        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[B_pos] = A[A_pos];
    }
}

__global__ void crop_scale_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor, int mode, float constant, float* rnd) {
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;

        // Compute random coordinates
        float scale = ((factor[1]-factor[0]) * rnd[b] + factor[0]);
        int w = irows * scale;
        int h = icols * scale;
        int x = (int)((icols-w) * rnd[b+2]);
        int y = (int)((irows-h) * rnd[b+3]);

        int coords_from_x = x;
        int coords_to_x = x+w;
        int coords_from_y = y;
        int coords_to_y = y+h;

        int A_hc = coords_to_y-coords_from_y+1;
        int A_wc = coords_to_x-coords_from_x+1;

        // Interpolate indices
        if(mode==2) { // Nearest
            int Ai = (Bi * A_hc) / orows + coords_from_y;
            int Aj = (Bj * A_wc) / ocols + coords_from_x;

            int A_pos = b * A_stride[0] + c * A_stride[1] + Ai * A_stride[2] + Aj * A_stride[3];
            int B_pos = b * B_stride[0] + c * B_stride[1] + Bi * B_stride[2] + Bj * B_stride[3];

            B[B_pos] = A[A_pos];
        }
    }
}

__global__ void cutout_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor_x, float* factor_y, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int offsets[2] = {0, 0};

        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;
        // printf("A={%d, %d, %d, %d}\n", b, c, Ai, Aj);
        // printf("B={%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // Compute random coordinates
        int w = (int)(icols * ((factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]));
        int h = (int)(irows * ((factor_y[1]-factor_y[0]) * rnd[b+1] + factor_y[0]));
        int x = (int)((icols-w) * rnd[b]);
        int y = (int)((irows-h) * rnd[b+1]);

        int coords_from_x = x;
        int coords_to_x = x+w;
        int coords_from_y = y;
        int coords_to_y = y+h;

        // Compute coordinates
        int Ai = Bi + offsets[0];  // Start from the (0,0) of the cropping area
        int Aj = Bj + offsets[1];

        bool inRegion = Ai >= coords_from_y && Ai <= coords_to_y && Aj >= coords_from_x && Aj <= coords_to_x;
        int B_pos = b*B_stride[0] + c*B_stride[1] + Bi*B_stride[2] + Bj*B_stride[3];  // We always walk through the whole B tensor

        if (inRegion){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[B_pos] = A[A_pos];
        }else{
            B[B_pos] = constant;
        }

    }
}