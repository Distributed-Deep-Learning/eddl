#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.1
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"


__global__ void shift(float* A, float* B, int batch, int depth, int irows, int icols, int* shift, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        int Ai = Bi - shift[0];
        int Aj = Bj - shift[1];

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            if(mode==0){ // constant
                B[thread_id_x] = constant;
            }
        }
    }

}

__global__ void rotate(float* A, float* B, int batch, int depth, int irows, int icols, float angle, int* axis, bool reshape, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    // Not implemented
    if (thread_id_x < ops){
        B[thread_id_x] = constant;
    }
}

__global__ void scale(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* new_shape, int mode, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*orows*ocols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // Interpolate indices
        int Ai = (Bi * irows) / new_shape[0];
        int Aj = (Bj * icols) / new_shape[1];

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            if(mode==0){ // constant
                B[thread_id_x] = constant;
            }
        }
    }

}

__global__ void flip(float* A, float* B, int batch, int depth, int irows, int icols, int axis){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        int pos[2] = {Bi, Bj}; 
        if(axis+2==2){ pos[axis] = (irows-1) - pos[axis]; }
        else if(axis+2==3){ pos[axis] = (icols-1) - pos[axis]; }

        int Ai = pos[0]; 
        int Aj = pos[1];

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }
    }
}


__global__ void crop(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, float constant){
   long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
   long int ops = batch * depth*irows*icols;

   if (thread_id_x < ops){
       int offsets[2] = {0, 0};
       offsets[0] = irows/2.0f - orows/2.0f+1;
       offsets[1] = icols/2.0f - ocols/2.0f+1;

       int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
       int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

       //--------------
       int b = thread_id_x / B_stride[0] % batch;
       int c = thread_id_x / B_stride[1] % depth;
       int Bi = thread_id_x / B_stride[2] % orows;
       int Bj = thread_id_x / B_stride[3] % ocols;

       int Ai = Bi;
       int Aj = Bj;
       if(irows!=orows) { Ai+= coords_from[0]; }
       if(icols!=ocols) { Aj+= coords_from[1]; }
       //--------------
       // printf("A={%d, %d, %d, %d}\n", b, c, Ai, Aj);
       // printf("B={%d, %d, %d, %d}\n", b, c, Bi, Bj);

       // Due to the offsets
       int B_pos = b*B_stride[0] + c*B_stride[1] + (Bi-offsets[0])*B_stride[2] + (Bj-offsets[1])*B_stride[3];
       if (Ai >= coords_from[0] && Ai <= coords_to[0] && Aj >= coords_from[1] && Aj <= coords_to[1]){
           int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
           B[B_pos] = A[A_pos];
       }else{
          B[B_pos] = constant;
      }

       
   }
}


__global__ void crop_scale(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, int* coords_from, int* coords_to, float constant){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_wc = coords_to[0]-coords_from[0]+1;
        int A_hc = coords_to[0]-coords_from[1]+1;

        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;

        // Interpolate indices
        int Ai = (Bi * A_hc) / orows + coords_from[0];
        int Aj = (Bj * A_wc) / ocols + coords_from[1];

        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[thread_id_x] = A[A_pos];
    }
}

__global__ void cutout(float* A, float* B, int batch, int depth, int irows, int icols, int* coords_from, int* coords_to, float constant){
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
  long int ops = batch*depth*irows*icols;

  if (thread_id_x < ops){
      int B_stride[4] = {depth*irows*icols, irows*icols, icols, 1};

      //--------------
      int b = thread_id_x / B_stride[0] % batch;
      int c = thread_id_x / B_stride[1] % depth;
      int Bi = thread_id_x / B_stride[2] % irows;
      int Bj = thread_id_x / B_stride[3] % icols;
      //--------------
       //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

      if (Bi >= coords_from[0] && Bi <= coords_to[0] && Bj >= coords_from[1] && Bj <= coords_to[1]){
          B[thread_id_x] = constant;
      }else{
          B[thread_id_x] = A[thread_id_x];
      }

  }
}


__global__ void shift_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor_x, float* factor_y, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        int shift_x = (int)(icols * ((factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]));
        int shift_y = (int)(irows * ((factor_y[1]-factor_y[0]) * rnd[b+1] + factor_y[0]));

        int Ai = Bi - shift_x;
        int Aj = Bj - shift_y;

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            if(mode==0){ // constant
                B[thread_id_x] = constant;
            }
        }
    }

}

__global__ void rotate_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor, int* axis, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    // TODO: Implement
    if (thread_id_x < ops){
        B[thread_id_x] = constant;
    }
}

__global__ void scale_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor, int mode, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*orows*ocols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // TODO: Center image
        float scale = (factor[1]-factor[0]) * rnd[b] + factor[0];
        int new_shape_x = (int)(icols * scale);
        int new_shape_y = (int)(irows * scale);
        
        // Interpolate indices
        int Ai = (Bi * irows) / orows;
        int Aj = (Bj * icols) / ocols;

        if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            if(mode==0){ // constant
                B[thread_id_x] = constant;
            }
        }
    }

}

__global__ void flip_random(float* A, float* B, int batch, int depth, int irows, int icols, int axis, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int *B_stride = A_stride;

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        bool apply = rnd[b] >= 0.5f;
        if (apply){
            int pos[2] = {Bi, Bj};
            if(axis+2==2){ pos[axis] = (irows-1) - pos[axis]; }
            else if(axis+2==3){ pos[axis] = (icols-1) - pos[axis]; }

            int Ai = pos[0];
            int Aj = pos[1];

            if (Ai >= 0 && Ai < irows && Aj >= 0 && Aj < icols){
                int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
                B[thread_id_x] = A[A_pos];
            }
        }else{
            B[thread_id_x] = A[thread_id_x];
        }
        
    }
}


__global__ void crop_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor_x, float* factor_y, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
      
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;

        //--------------
        // printf("A={%d, %d, %d, %d}\n", b, c, Ai, Aj);
        // printf("B={%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // Performs a crop with padding
        int offsets[2] = {0, 0};
        offsets[0] = irows/2.0f - orows/2.0f+1;
        offsets[1] = icols/2.0f - ocols/2.0f+1;

        // Compute random coordinates
        int x1 = (int)(icols * (factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]);
        int x2 = (int)(icols * (factor_x[1]-factor_x[0]) * rnd[b+1] + factor_x[0]);
        int y1 = (int)(irows * (factor_y[1]-factor_y[0]) * rnd[b+2] + factor_y[0]);
        int y2 = (int)(irows * (factor_y[1]-factor_y[0]) * rnd[b+3] + factor_y[0]);

        int coords_from_x = min(x1, x2);
        int coords_to_x = max(x1, x2);
        int coords_from_y = min(y1, y2);
        int coords_to_y = max(y1, y2);

        int Ai = Bi;  
        int Aj = Bj;
        if(irows!=orows) { Ai+= coords_from_x; }
        if(icols!=ocols) { Aj+= coords_from_y; }


        if (Ai >= coords_from_y && Ai <= coords_to_y && Aj >= coords_from_x && Aj <= coords_to_x){
            int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
            B[thread_id_x] = A[A_pos];
        }else{
            B[thread_id_x] = constant;
        }


    }
}


__global__ void crop_scale_random(float* A, float* B, int batch, int depth, int irows, int icols, int orows, int ocols, float* factor_x, float* factor_y, float constant, float* rnd) {
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch * depth*irows*icols;

    if (thread_id_x < ops){
        int A_stride[4] = {depth*irows*icols, irows*icols, icols, 1};
        int B_stride[4] = {depth*orows*ocols, orows*ocols, ocols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % orows;
        int Bj = thread_id_x / B_stride[3] % ocols;

        // Compute random coordinates
        int x1 = (int)(icols * (factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]);
        int x2 = (int)(icols * (factor_x[1]-factor_x[0]) * rnd[b+1] + factor_x[0]);
        int y1 = (int)(irows * (factor_y[1]-factor_y[0]) * rnd[b+2] + factor_y[0]);
        int y2 = (int)(irows * (factor_y[1]-factor_y[0]) * rnd[b+3] + factor_y[0]);

        int coords_from_x = min(x1, x2);
        int coords_to_x = max(x1, x2);
        int coords_from_y = min(y1, y2);
        int coords_to_y = max(y1, y2);
        
        int A_hc = coords_to_y-coords_from_y+1;
        int A_wc = coords_to_x-coords_from_x+1;

        // Interpolate indices
        int Ai = (Bi * A_hc) / orows + coords_from_x;
        int Aj = (Bj * A_wc) / ocols + coords_from_y;

        int A_pos = b*A_stride[0] + c*A_stride[1] + Ai*A_stride[2] + Aj*A_stride[3];
        B[thread_id_x] = A[A_pos];
    }
}

__global__ void cutout_random(float* A, float* B, int batch, int depth, int irows, int icols, float* factor_x, float* factor_y, float constant, float* rnd){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    long int ops = batch*depth*irows*icols;

    if (thread_id_x < ops){
        int B_stride[4] = {depth*irows*icols, irows*icols, icols, 1};

        //--------------
        int b = thread_id_x / B_stride[0] % batch;
        int c = thread_id_x / B_stride[1] % depth;
        int Bi = thread_id_x / B_stride[2] % irows;
        int Bj = thread_id_x / B_stride[3] % icols;
        //--------------
        //printf("{%d, %d, %d, %d}\n", b, c, Bi, Bj);

        // Compute random coordinates
        int x1 = (int)(icols * (factor_x[1]-factor_x[0]) * rnd[b] + factor_x[0]);
        int x2 = (int)(icols * (factor_x[1]-factor_x[0]) * rnd[b+1] + factor_x[0]);
        int y1 = (int)(irows * (factor_y[1]-factor_y[0]) * rnd[b+2] + factor_y[0]);
        int y2 = (int)(irows * (factor_y[1]-factor_y[0]) * rnd[b+3] + factor_y[0]);

        int coords_from_x = min(x1, x2);
        int coords_to_x = max(x1, x2);
        int coords_from_y = min(y1, y2);
        int coords_to_y = max(y1, y2);
        
        if (Bi >= coords_from_y && Bi <= coords_to_y && Bj >= coords_from_x && Bj <= coords_to_x){
            B[thread_id_x] = constant;
        }else{
            B[thread_id_x] = A[thread_id_x];
        }

    }
}