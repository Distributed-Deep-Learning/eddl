#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"



void gpu_reduction(ReduceDescriptor *RD){
  int device=RD->I->gpu_device;

  hipSetDevice(device);

  int i,j,d,s,max,p;


  // [MEAN]: Compute items to be reduced
  if (RD->m==0) {
      d=1;
      for(i=0;i<RD->axis.size();i++){
          d *= RD->I->shape[RD->axis[i]];
      }
  }

  if (RD->ind==nullptr) {
    fprintf(stderr,"Mem GPU ind\n");

    RD->max=0;
    for(i=0;i<RD->index.size();i++)
       if(RD->max<RD->index[i].size()) RD->max=RD->index[i].size();
    RD->max++;
    s=RD->index.size()*RD->max;

    int *ind=(int *)malloc(s*sizeof(int));

    for(i=0;i<s;i++) ind[i]=-1;

    for(i=0;i<RD->index.size();i++) {
      p=i*max;
      for(j=0;j<RD->index[i].size();j++,p++)
        ind[p]=RD->index[i][j];
    }

    check_cuda(hipMalloc((void**)&(RD->ind),s*sizeof(int)),"create_index");
    check_cuda(hipMemcpy(ind,RD->ind,s*sizeof(int),hipMemcpyHostToDevice),"copy ind");

    free(ind);
  }

  //reduce
  dim3 dimGrid(RD->index.size());
  dim3 dimBlock(1);

  reduction_kernel<<<dimGrid,dimBlock>>>(RD->I->ptr, RD->O->ptr, RD->S->ptr,RD->m, RD->keepdims,d,RD->ind,RD->max);
  check_cuda(hipDeviceSynchronize(), "reduction_kernel");

}





void gpu_reduction_back(ReduceDescriptor *RD){
  int device=RD->I->gpu_device;

  hipSetDevice(device);

  float val,sum;
  int ind;
  int d,i;


  // [MEAN]: Compute items to be reduced
  if (RD->m==0) {
      d=1;
      for(i=0;i<RD->axis.size();i++){
          d *= RD->I->shape[RD->axis[i]];
      }
  }

  //reduce
  //dim3 dimGrid(RD->index.size());
  //dim3 dimBlock(1024);

  //reduction_back_kernel<<<dimGrid,dimBlock>>>(RD->I->ptr, RD->O->ptr, RD->m, RD->keepdims);
  //check_cuda(hipDeviceSynchronize(), "reduction_back_kernel");
}
