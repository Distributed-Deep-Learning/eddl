#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "tensor_cuda.h"
#include "tensor_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"
