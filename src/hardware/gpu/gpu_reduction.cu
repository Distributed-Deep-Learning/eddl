#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.1
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include <thrust/device_ptr.h>
//#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"



void gpu_reduction(ReduceDescriptor *RD){
  int device=RD->I->gpu_device;

  hipSetDevice(device);

  int i,j,d,s,p;


  // [MEAN]: Compute items to be reduced
  if (RD->m==0) {
      d=1;
      for(i=0;i<RD->axis.size();i++){
          d *= RD->I->shape[RD->axis[i]];
      }
  }

  //////// Init
  if (RD->ind==nullptr) {
    RD->red_size=RD->index[0].size();
    s=RD->index.size()*RD->red_size;

    int *ind=(int *)malloc(s*sizeof(int));

    for(i=0;i<RD->index.size();i++) {
      p=i*RD->red_size;
      for(j=0;j<RD->index[i].size();j++,p++)
        ind[p]=RD->index[i][j];
    }

    if (RD->m<2) RD->S=RD->O;

    check_cuda(hipMalloc((void**)&(RD->ind),s*sizeof(int)),"create_index");
    check_cuda(hipDeviceSynchronize(), "create ind");

    check_cuda(hipMemcpy(RD->ind,ind,s*sizeof(int),hipMemcpyHostToDevice),"copy ind");
    check_cuda(hipDeviceSynchronize(), "copy");

    check_cuda(hipMalloc((void**)&(RD->red),RD->index.size()*sizeof(float)),"create_tensor");

    free(ind);
  }
  /////////////

  int fast=0;
  if (RD->factor*RD->index.size()<RD->red_size) fast=1;


  if ((fast)&&((RD->m==0)&&(RD->keepdims))) {//mean with keepdims=true (BN)

    setDims(RD->O);
    reduction_permute<<<dimGrid,dimBlock>>>(RD->I->ptr, RD->O->ptr, RD->ind, RD->O->size);
    check_cuda(hipDeviceSynchronize(), "reduction_kernel");

    for(int i=0;i<RD->index.size();i++) {
      float *ptr=RD->O->ptr+(i*RD->red_size);

      thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(ptr);
      thrust::device_ptr<float> base = thrust::device_pointer_cast(RD->red);

      float sum = thrust::reduce(dev_ptr, dev_ptr + RD->red_size);
      thrust::fill(base + i, base + i + 1, (float)sum/RD->red_size);
    }

    reduction_kernel_keep<<<dimGrid,dimBlock>>>(RD->red, RD->O->ptr,RD->ind, RD->index.size(),RD->red_size);
    check_cuda(hipDeviceSynchronize(), "reduction_kernel");

  }else{ // still slow for max, min on conv
    RD->O->set(0.0);
    dim3 dimGrid(RD->index.size());
    dim3 dimBlock(1);
    reduction_kernel<<<dimGrid,dimBlock>>>(RD->I->ptr, RD->O->ptr, RD->S->ptr,RD->m, RD->keepdims,d,RD->ind,RD->red_size);
    check_cuda(hipDeviceSynchronize(), "reduction_kernel");
  }

}


//////
////// back
//////
void gpu_reduction_back(ReduceDescriptor *RD){
  int device=RD->I->gpu_device;

  hipSetDevice(device);

  int d,i;

  // [MEAN]: Compute items to be reduced
  if (RD->m==0) {
      d=1;
      for(i=0;i<RD->axis.size();i++){
          d *= RD->I->shape[RD->axis[i]];
      }
  }

  int fast=0;
  if (RD->factor*RD->index.size()<RD->red_size) fast=1;

  if ((fast)&&((RD->m==0)&&(RD->keepdims))) {// mean with keepdims=true (BN)
    setDims(RD->ID);
    reduction_permute<<<dimGrid,dimBlock>>>(RD->D->ptr, RD->ID->ptr, RD->ind, RD->O->size);
    check_cuda(hipDeviceSynchronize(), "reduction_kernel");

    for(int i=0;i<RD->index.size();i++) {
      float *ptr=RD->ID->ptr+(i*RD->red_size);

      thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(ptr);
      thrust::device_ptr<float> base = thrust::device_pointer_cast(RD->red);

      float sum = thrust::reduce(dev_ptr, dev_ptr + RD->red_size);
      thrust::fill(base+i, base + i + 1, (float)sum/RD->red_size);
    }

    reduction_kernel_keep<<<dimGrid,dimBlock>>>(RD->red, RD->ID->ptr, RD->ind, RD->index.size(),RD->red_size);
    check_cuda(hipDeviceSynchronize(), "reduction_kernel");

  }else{ // still slow for max, min on conv
    dim3 dimGrid(RD->index.size());
    dim3 dimBlock(1);
    reduction_back_kernel<<<dimGrid,dimBlock>>>(RD->D->ptr, RD->ID->ptr, RD->S->ptr,RD->m, RD->keepdims,d,RD->ind,RD->red_size);
    check_cuda(hipDeviceSynchronize(), "reduction_kernel");
  }
}
