#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"

void gpu_range(Tensor *A, float min, float step, int size) {
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    //range<<<dimGrid,dimBlock>>>(A->ptr, r, c, min, step, size);
    check_cuda(hipDeviceSynchronize(),"range");
}
