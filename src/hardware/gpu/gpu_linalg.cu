#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.7
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: April 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include <thrust/device_ptr.h>
//#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"
#include "eddl/hardware/gpu/gpu_hw.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"



// square<T> computes the square of a number f(x) -> x*x
template <typename T>
struct sqr
{
    __host__ __device__
    T operator()(const T& x) const {
        return x * x;
    }
};

float gpu_norm(Tensor *A, string ord){
    int device=A->gpu_device;
    hipSetDevice(device);

    if (ord=="fro"){
        thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(A->ptr);

        // setup arguments
        sqr<float>        unary_op;
        thrust::plus<float> binary_op;
        float init = 0;

        float abs_sum_sqr = thrust::transform_reduce(dev_ptr, dev_ptr + A->size, unary_op, init, binary_op);
        float norm = std::sqrt(abs_sum_sqr);

        return norm;
    }else{
        msg("Not yet implemented", "cpu_norm");
    }

    return 0.0f;
}

