/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.3
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


int* get_block_dim(int N, int blockSize){
    int* res = new int[2];
    int blocks = (N + blockSize - 1) / blockSize;
    if (N<blockSize) { blockSize = N; }

    res[0] = blocks;
    res[1] = blockSize;
    return res;
}

void copy_cpu2gpu(void * cpu_addresses, void* gpu_addresses, int size, bool delete_cpu){
    check_cuda(hipMalloc((void**)&(gpu_addresses), size), "create address mapping");
    check_cuda(hipDeviceSynchronize(), "create");


    check_cuda(hipMemcpy(gpu_addresses, cpu_addresses, size, hipMemcpyHostToDevice), "copy address mapping");
    check_cuda(hipDeviceSynchronize(), "copy");

    // Free CPU pointer?
    if (delete_cpu) { delete cpu_addresses; }
}

void gpu_copy_to_gpu(float *nptr,Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);
    check_cuda(hipMemcpy(A->ptr,nptr,A->size*sizeof(float),hipMemcpyHostToDevice),"gpu_copy_to_gpu");
}


void gpu_copy_from_gpu(Tensor *A,float *nptr){
    int device=A->gpu_device;
    hipSetDevice(device);
    check_cuda(hipMemcpy(nptr,A->ptr,A->size*sizeof(float),hipMemcpyDeviceToHost),"gpu_copy_to_gpu");
}


void gpu_copy_gpu(Tensor *A,Tensor *B){
    int device=A->gpu_device;
    hipSetDevice(device);
    check_cuda(hipMemcpy(B->ptr,A->ptr,A->size*sizeof(float),hipMemcpyDeviceToDevice),"gpu_copy_gpu");
}


void gpu_fill(Tensor *A,int aini,int aend,Tensor *B,int bini,int bend,int inc){
    int device=A->gpu_device;
    hipSetDevice(device);

    int at=A->size/A->shape[0];
    int bt=B->size/B->shape[0];

    int t=1;
    for(int i=2;i<B->ndim;i++)
        t*=B->shape[i];

    int tot=B->shape[0]*(bend-1)*B->shape[1]*t;
    int r,c;

    while (aend-aini>0) {

        if ((aend-aini)>MAX_TPB) r=MAX_TPB;
        else r=(aend-aini);
        c=t;

        dim3 dimGrid(A->shape[0],c);
        dim3 dimBlock(r);

        fill<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,t,aini,at,bini,bt,tot,inc);
        aini+=MAX_TPB;
        bini+=MAX_TPB;

    }

    //check_cuda(hipDeviceSynchronize(),"fill");

}


void gpu_mask(Tensor *A,float v) {

    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    mask<<<dimGrid,dimBlock>>>(A->ptr,v,A->size);
    check_cuda(hipDeviceSynchronize(),"mask");

}


void gpu_fill_(Tensor *A, float v) {
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    fill_<<<dimGrid,dimBlock>>>(A->ptr,v,A->size);
    check_cuda(hipDeviceSynchronize(),"set");
}


void gpu_select(Tensor *A, Tensor *B, SelDescriptor *sd){
    int device=A->gpu_device;
    hipSetDevice(device);

     if(sd->gpu_addresses == nullptr){
        // copy_cpu2gpu(sd->cpu_addresses, sd->gpu_addresses, B->size*sizeof(int), true);

        check_cuda(hipMalloc((void**)&(sd->gpu_addresses), B->size*sizeof(int)), "create address mapping");
        check_cuda(hipDeviceSynchronize(), "create");

        check_cuda(hipMemcpy(sd->gpu_addresses, sd->cpu_addresses, B->size*sizeof(int), hipMemcpyHostToDevice), "copy address mapping");
        check_cuda(hipDeviceSynchronize(), "copy");
    }


    setDims(B);  // B is the small
    select<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, B->size, sd->gpu_addresses);
    check_cuda(hipDeviceSynchronize(), "select");
}

void gpu_select_back(Tensor *A, Tensor *B, SelDescriptor *sd){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy indices from host to device
    if(sd->gpu_addresses == nullptr){
        // copy_cpu2gpu(sd->cpu_addresses, sd->gpu_addresses, A->size*sizeof(int), true);

        check_cuda(hipMalloc((void**)&(sd->gpu_addresses), A->size*sizeof(int)), "create address mapping");
        check_cuda(hipDeviceSynchronize(), "create");

        check_cuda(hipMemcpy(sd->gpu_addresses, sd->cpu_addresses, A->size*sizeof(int), hipMemcpyHostToDevice), "copy address mapping");
        check_cuda(hipDeviceSynchronize(), "copy");
    }


    setDims(A);  // A is the small
    select_back<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size, sd->gpu_addresses);
    check_cuda(hipDeviceSynchronize(), "select_back");
}


void gpu_set_select(Tensor *A, Tensor *B, SelDescriptor *sd){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy indices from host to device
    if(sd->gpu_addresses == nullptr){
        // copy_cpu2gpu(sd->cpu_addresses, sd->gpu_addresses, B->size*sizeof(int), true);

        check_cuda(hipMalloc((void**)&(sd->gpu_addresses), B->size*sizeof(int)), "create address mapping");
        check_cuda(hipDeviceSynchronize(), "create");

        check_cuda(hipMemcpy(sd->gpu_addresses, sd->cpu_addresses, B->size*sizeof(int), hipMemcpyHostToDevice), "copy address mapping");
        check_cuda(hipDeviceSynchronize(), "copy");
    }

    setDims(B);  // B is the small
    set_select<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, B->size, sd->gpu_addresses);
    check_cuda(hipDeviceSynchronize(), "set_select");
}


void gpu_set_select_back(Tensor *A, Tensor *B, SelDescriptor *sd){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Copy indices from host to device
    if(sd->gpu_addresses == nullptr){
        // copy_cpu2gpu(sd->cpu_addresses, sd->gpu_addresses, B->size*sizeof(int), true);

        check_cuda(hipMalloc((void**)&(sd->gpu_addresses), B->size*sizeof(int)), "create address mapping");
        check_cuda(hipDeviceSynchronize(), "create");

        check_cuda(hipMemcpy(sd->gpu_addresses, sd->cpu_addresses, B->size*sizeof(int), hipMemcpyHostToDevice), "copy address mapping");
        check_cuda(hipDeviceSynchronize(), "copy");
    }

    setDims(B);  // B is the small
    set_select_back<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, B->size, sd->gpu_addresses);
    check_cuda(hipDeviceSynchronize(), "set_select_back");
}

void gpu_concat(Tensor *A, vector<Tensor*> t, unsigned int axis, bool derivative){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Walk through all the tensors to concat one axis (once)
    //unsigned int offset = 0;
    unsigned int size = 0;
    int steps = A->stride[axis] * A->shape[axis];  // Equivalent to A->stride[axis-1], but without the negative index problem

    // Walk through each tensor
    #pragma omp parallel for
    for (unsigned int i = 0; i < t.size(); i++) {
        int offset = i*size;
        size = t[i]->stride[axis] * t[i]->shape[axis];

        // Copy n bytes from src to dest
        float *dest = A->ptr + offset;
        float *src = t[i]->ptr;


        setDims(t[i]);
        concat<<<dimGrid,dimBlock>>>(dest, src, t[i]->size, size, steps, derivative);
        check_cuda(hipDeviceSynchronize(),"gpu_concat");

    }
}
