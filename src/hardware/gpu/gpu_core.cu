/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.2
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


int* get_block_dim(int N, int blockSize){
  int* res = new int[2];
  int blocks = (N + blockSize - 1) / blockSize;
  if (N<blockSize) { blockSize = N; }

   res[0] = blocks;
   res[1] = blockSize;
  return res;
}


void gpu_copy_to_gpu(float *nptr,Tensor *A){
  int device=A->gpu_device;
  hipSetDevice(device);
  check_cuda(hipMemcpy(A->ptr,nptr,A->size*sizeof(float),hipMemcpyHostToDevice),"gpu_copy_to_gpu");
}


void gpu_copy_from_gpu(Tensor *A,float *nptr){
  int device=A->gpu_device;
  hipSetDevice(device);
  check_cuda(hipMemcpy(nptr,A->ptr,A->size*sizeof(float),hipMemcpyDeviceToHost),"gpu_copy_to_gpu");
}


void gpu_copy_gpu(Tensor *A,Tensor *B){
  int device=A->gpu_device;
  hipSetDevice(device);
  check_cuda(hipMemcpy(B->ptr,A->ptr,A->size*sizeof(float),hipMemcpyDeviceToDevice),"gpu_copy_gpu");
}


void gpu_fill(Tensor *A,int aini,int aend,Tensor *B,int bini,int bend,int inc){
  int device=A->gpu_device;
  hipSetDevice(device);

  int at=A->size/A->shape[0];
  int bt=B->size/B->shape[0];

  int t=1;
  for(int i=2;i<B->ndim;i++)
    t*=B->shape[i];

  int tot=B->shape[0]*(bend-1)*B->shape[1]*t;
  int r,c;

  while (aend-aini>0) {

      if ((aend-aini)>MAX_TPB) r=MAX_TPB;
      else r=(aend-aini);
      c=t;

      dim3 dimGrid(A->shape[0],c);
      dim3 dimBlock(r);

      fill<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,t,aini,at,bini,bt,tot,inc);
      aini+=MAX_TPB;
      bini+=MAX_TPB;

  }

    //check_cuda(hipDeviceSynchronize(),"fill");

}


void gpu_mask(Tensor *A,float v) {

  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  mask<<<dimGrid,dimBlock>>>(A->ptr,v,A->size);
  check_cuda(hipDeviceSynchronize(),"mask");

}


void gpu_select(Tensor *A, Tensor *B, vector<vector<int>> indices){
    int device=A->gpu_device;
    hipSetDevice(device);

    // Prepare indices
    int* cpu_min_indices = new int[4];
    for(int i=0; i<indices.size(); i++){
        cpu_min_indices[i] = indices[i][0];
    }

    // Copy indices from host to device
    int *gpu_min_indices; hipMalloc((int**)&gpu_min_indices, 4*sizeof(int));
    hipMemcpy(gpu_min_indices, cpu_min_indices, 4*sizeof(int), hipMemcpyHostToDevice);

    setDims(B);
    select<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, B->shape[0], B->shape[1], B->shape[2], B->shape[3], gpu_min_indices);
    check_cuda(hipDeviceSynchronize(), "select");
}

void gpu_fill_(Tensor *A, float v) {
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    fill_<<<dimGrid,dimBlock>>>(A->ptr,v,A->size);
    check_cuda(hipDeviceSynchronize(),"set");
}
