/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.2
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


bool gpu_allclose(Tensor *A, Tensor *B, float rtol, float atol, bool equal_nan){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    bool close = true;
    allclose<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size, close);
    check_cuda(hipDeviceSynchronize(), "allclose");
    return close
}

void gpu_isclose(Tensor *A, Tensor *B, Tensor *C, float rtol, float atol, bool equal_nan){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    isclose<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "isclose");
}

void gpu_greater(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    greater<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "greater");
}

void gpu_greater_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    greater_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "greater_equal");
}

void gpu_less(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    less<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "less");
}

void gpu_less_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    less_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "less_equal");
}

void gpu_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "equal");
}

void gpu_not_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    not_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "not_equal");
}
