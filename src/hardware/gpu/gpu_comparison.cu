#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "tensor_cuda.h"
#include "tensor_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


// MAX THREADS PER BLOCK
#define MAX_TPB 1024
#define setDims(A) int r,c;r=(A->size/MAX_TPB);if (r==0) {r=1;c=A->size;}else {if (A->size%MAX_TPB) r++;c=MAX_TPB;}dim3 dimGrid(r);dim3 dimBlock(c);

extern hipblasHandle_t hcublas[64];
extern hiprandGenerator_t random_generator[64];