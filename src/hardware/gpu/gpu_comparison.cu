/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.7
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: April 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "eddl/hardware/gpu/gpu_tensor.h"
#include "eddl/hardware/gpu/gpu_kernels.h"
#include "eddl/hardware/gpu/gpu_hw.h"

#include "eddl/tensor/tensor.h"
#include "eddl/descriptors/descriptors.h"


// CPU: Logic functions: Comparisons
void gpu_isfinite(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isfinite<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isfinite");
}

void gpu_isinf(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isinf<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isinf");
}

void gpu_isnan(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isnan<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isnan");
}

void gpu_isneginf(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isneginf<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isneginf");
}

void gpu_isposinf(Tensor *A, Tensor* B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_isposinf<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "gpu_isposinf");
}


// CPU: Logic functions: Comparisons
void gpu_logical_and(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_and<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_and");
}

void gpu_logical_or(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_or<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_or");
}

void gpu_logical_not(Tensor *A, Tensor *B){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_not<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_not");
}

void gpu_logical_xor(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_xor<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "logical_xor");
}

// GPU: Logic functions: Truth value testing
bool gpu_all(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    bool result = true;
    gpu_logical_all<<<dimGrid,dimBlock>>>(A->ptr, A->size, result);
    check_cuda(hipDeviceSynchronize(), "all");
    return result;
}

bool gpu_any(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    bool result = false;
    gpu_logical_any<<<dimGrid,dimBlock>>>(A->ptr, A->size, result);
    check_cuda(hipDeviceSynchronize(), "any");
    return result;
}

bool gpu_allclose(Tensor *A, Tensor *B, float rtol, float atol, bool equal_nan){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    bool close = true;
    gpu_logical_allclose<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, A->size, rtol, atol, equal_nan, close);
    check_cuda(hipDeviceSynchronize(), "allclose");
    return close;
}

void gpu_isclose(Tensor *A, Tensor *B, Tensor *C, float rtol, float atol, bool equal_nan){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_logical_isclose<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, rtol, atol, equal_nan, A->size);
    check_cuda(hipDeviceSynchronize(), "isclose");
}

void gpu_greater(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "greater");
}

void gpu_greater(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "greater");
}

void gpu_greater_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "greater_equal");
}

void gpu_greater_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_greater_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "greater_equal");
}

void gpu_less(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "less");
}

void gpu_less(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "less");
}

void gpu_less_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "less_equal");
}

void gpu_less_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_less_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "less_equal");
}

void gpu_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "equal");
}

void gpu_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "equal");
}

void gpu_not_equal(Tensor *A, Tensor *B, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_not_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, v, A->size);
    check_cuda(hipDeviceSynchronize(), "not_equal");
}

void gpu_not_equal(Tensor *A, Tensor *B, Tensor *C){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    gpu_not_equal<<<dimGrid,dimBlock>>>(A->ptr, B->ptr, C->ptr, A->size);
    check_cuda(hipDeviceSynchronize(), "not_equal");
}