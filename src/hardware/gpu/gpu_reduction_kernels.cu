#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.5
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: April 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include "eddl/hardware/gpu/gpu_kernels.h"


__global__ void reduce_mean(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    atomicAdd(&(B[map[thread_id_x]]),A[thread_id_x]);
  }

}

__global__ void reduce_op_sum(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]+=B[map[thread_id_x]];
  }

}
__global__ void reduce_op_diff(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]-=B[map[thread_id_x]];
  }

}
__global__ void reduce_op_mult(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]*=B[map[thread_id_x]];
  }

}
__global__ void reduce_op_div(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]/=B[map[thread_id_x]];
  }

}


//dim3 dimGrid(RD->index.size());
//dim3 dimBlock(1);
__global__ void reduction_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int rs)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  int j;
  float sum=0;
  float v,val;

  int i;

  int p=rs*blockIdx.x;


  for(j=0;j<rs;j++,p++) {
      v=I[ind[p]];
      if (m==2) {
          if (j==0) {val=v;i=p;}
          else if (v>val) {
              val=v;
              i=p;
          }
      }
      else if (m==3) {
        if (j==0) {val=v;i=p;}
        else if (v<val) {
            val=v;
            i=p;
        }
      }
      else sum+=v;
  }

  p=rs*blockIdx.x;
  // set in Output
  if (m<2) { // mean or sum
      if (m==0) sum/=d;
      if (keepdims) {
        for(j=0;j<rs;j++,p++)
            O[ind[p]]=sum;
      }
      else O[thread_id_x]=sum;
  }
  else { // rs or min
      if (keepdims) {
        for(j=0;j<rs;j++,p++) {
              O[ind[p]]=val;
              S[ind[p]]=i;
          }
      }
      else {
          O[thread_id_x]=val;
          S[thread_id_x]=i;
      }
  }

}


//dim3 dimGrid(RD->index.size());
//dim3 dimBlock(1);
__global__ void reduction_back_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int rs)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    int j;
    float val=0;
    int p;

    // set in Delta
    if (m>=2) {
      int p=S[thread_id_x];
      O[p]+=I[thread_id_x];
    }
    else {
      p=rs*blockIdx.x;
      if(keepdims) {
        for(j=0;j<rs;j++,p++)
          val+=I[ind[p]];
      }
      else val=I[thread_id_x];
      if (m==0) val/=d;

      p=rs*blockIdx.x;
      for(j=0;j<rs;j++,p++)
        O[ind[p]]+=val;
    }
}



////////////////////
// FOR SUM and MEAN
// Faster in Conv
///////////////////

//dim3 dimGrid(red_size);
//dim3 dimBlock(RD->index.size());

__global__ void reduction_permute(float *I,float *O,int *ind,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size)
    O[thread_id_x]=I[ind[thread_id_x]];
}

__global__ void reduction_kernel_keep(float *red, float *O, int *ind, int size, int rsize)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    if (thread_id_x<size*rsize) {
        O[ind[thread_id_x]]=red[thread_id_x/rsize];
    }
}

__global__ void reduction_kernel_keep_inc(float *red, float *O, int *ind, int size, int rsize)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    if (thread_id_x<size*rsize) {
        O[ind[thread_id_x]]+=red[thread_id_x/rsize];
    }
}
