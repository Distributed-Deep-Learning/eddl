#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.1
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"
//dim3 dimGrid(RD->index.size());
//dim3 dimBlock(1);
__global__ void reduction_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int rs)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  int j;
  float sum=0;
  float v,val;

  int i;

  int p=rs*blockIdx.x;


  for(j=0;j<rs;j++,p++) {
      v=I[ind[p]];
      if (m==2) {
          if (j==0) {val=v;i=p;}
          else if (v>val) {
              val=v;
              i=p;
          }
      }
      else if (m==3) {
        if (j==0) {val=v;i=p;}
        else if (v<val) {
            val=v;
            i=p;
        }
      }
      else sum+=v;
  }

  p=rs*blockIdx.x;
  // set in Output
  if (m<2) { // mean or sum
      if (m==0) sum/=d;
      if (keepdims) {
        for(j=0;j<rs;j++,p++)
            O[ind[p]]=sum;
      }
      else O[thread_id_x]=sum;
  }
  else { // rs or min
      if (keepdims) {
        for(j=0;j<rs;j++,p++) {
              O[ind[p]]=val;
              S[ind[p]]=i;
          }
      }
      else {
          O[thread_id_x]=val;
          S[thread_id_x]=i;
      }
  }

}


//dim3 dimGrid(RD->index.size());
//dim3 dimBlock(1);
__global__ void reduction_back_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int rs)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    int j;
    float val=0;
    int p;

    // set in Delta
    if (m>=2) {
      int p=S[thread_id_x];
      O[p]+=I[thread_id_x];
    }
    else {
      p=rs*blockIdx.x;
      if(keepdims) {
        for(j=0;j<rs;j++,p++)
          val+=I[ind[p]];
      }
      else val=I[thread_id_x];
      if (m==0) val/=d;

      p=rs*blockIdx.x;
      for(j=0;j<rs;j++,p++)
        O[ind[p]]+=val;
    }
}



////////////////////
// FOR SUM and MEAN
// Faster in Conv
///////////////////

//dim3 dimGrid(red_size);
//dim3 dimBlock(RD->index.size());

__global__ void reduction_permute(float *I,float *O,int *ind,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size)
    O[thread_id_x]=I[ind[thread_id_x]];
}

__global__ void reduction_kernel_keep(float *red, float *O, int *ind, int size, int rsize)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    if (thread_id_x<size*rsize) {
        O[ind[thread_id_x]]=red[thread_id_x/rsize];
    }
}

__global__ void reduction_kernel_keep_inc(float *red, float *O, int *ind, int size, int rsize)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    if (thread_id_x<size*rsize) {
        O[ind[thread_id_x]]+=red[thread_id_x/rsize];
    }
}
