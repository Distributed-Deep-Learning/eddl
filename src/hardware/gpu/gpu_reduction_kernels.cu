#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.1
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"

__global__ void reduction_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int max)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  int j;
  float sum=0;
  float v,val;

  int i;

  int p=max*blockIdx.x;


  for(j=0;j<max && ind[p]!=-1;j++,p++) {
      v=I[ind[p]];
      if (m==2) {
          if (j==0) {val=v;i=p;}
          else if (v>val) {
              val=v;
              i=p;
          }
      }
      else if (m==3) {
        if (j==0) {val=v;i=p;}
        else if (v<val) {
            val=v;
            i=p;
        }
      }
      else sum+=v;
  }

  p=max*blockIdx.x;
  // set in Output
  if (m<2) { // mean or sum
      if (m==0) sum/=d;
      if (keepdims) {
        for(j=0;j<max&& ind[p]!=-1;j++,p++)
            O[ind[p]]=sum;
      }
      else O[thread_id_x]=sum;
  }
  else { // max or min
      if (keepdims) {
        for(j=0;j<max && ind[p]!=-1;j++,p++) {
              O[ind[p]]=val;
              S[ind[p]]=i;
          }
      }
      else {
          O[thread_id_x]=val;
          S[thread_id_x]=i;
      }
  }

}



__global__ void reduction_back_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int max)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    int j;
    float val=0;

    int p;

    // set in Delta
    if (m>=2) {
      int p=S[thread_id_x];
      O[p]+=I[thread_id_x];
    }
    else {
      p=max*blockIdx.x;
      if(keepdims) {
        for(j=0;j<max && ind[p]!=-1;j++,p++)
          val+=I[ind[p]];
      }
      else val=I[thread_id_x];
      if (m==0) val/=d;

      p=max*blockIdx.x;
      for(j=0;j<max && ind[p]!=-1;j++,p++)
        O[ind[p]]+=val;
    }
}
