#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.7
* copyright (c) 2020, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: April 2020
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

#include "eddl/hardware/gpu/gpu_kernels.h"

__global__ void gpu_max(float *A, float *B, int *map, int size, int size_reduction, bool argmax){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp_max = A[map[thread_id_x*size_reduction+0]];
        int tmp_argmax = 0;

        float val;
        for(int i=1; i<size_reduction; i++){
            val = A[map[thread_id_x*size_reduction+i]];
            if(val > tmp_max){
                tmp_max = val;
                tmp_argmax = i;
            }
        }
        
        // Choose if we're getting the maximum value or the position
        if(argmax) {
            B[thread_id_x] = (float)tmp_argmax;
        }else{
            B[thread_id_x] = tmp_max;
        }
    }
}

__global__ void gpu_min(float *A, float *B, int *map, int size, int size_reduction, bool argmin){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp_min = A[map[thread_id_x*size_reduction+0]];
        int tmp_argmin = 0;

        float val;
        for(int i=1; i<size_reduction; i++){
            val = A[map[thread_id_x*size_reduction+i]];
            if(val < tmp_min){
                tmp_min = val;
                tmp_argmin = i;
            }
        }

        // Choose if we're getting the minimum value or the position
        if(argmin) {
            B[thread_id_x] = (float)tmp_argmin;
        }else{
            B[thread_id_x] = tmp_min;
        }
    }
}

__global__ void gpu_sum(float *A, float *B, int *map, int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        atomicAdd(&B[map[thread_id_x]], A[thread_id_x]);
    }
}

__global__ void gpu_sum_abs(float *A, float *B, int *map, int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        atomicAdd(&B[map[thread_id_x]], abs(A[thread_id_x]));
    }
}

__global__ void gpu_prod(float *A, float *B, int *map, int size, int size_reduction){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp = 1.0f;
        for(int i=0; i<size_reduction; i++){
            tmp *= A[map[thread_id_x*size_reduction+i]];
        }

        B[thread_id_x] = tmp;
    }
}

__global__ void gpu_mean(float *A, float *B, int *map, int size, int size_reduction){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp = 0.0f;
        for(int i=0; i<size_reduction; i++){
            tmp += A[map[thread_id_x*size_reduction+i]];
        }

        B[thread_id_x] = tmp/(float)size_reduction;
    }
}

__global__ void gpu_var(float *A, float *B, int *map, int size, int size_reduction, bool unbiased){
    // IMPORTANT TRICK: B ALREADY CONTAINS THE MEAN!!!!!!!
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp;
        float sum = 0.0f;
        for(int i=0; i<size_reduction; i++){
            tmp = A[map[thread_id_x*size_reduction+i]] - B[thread_id_x];
            sum += tmp*tmp;
        }

        if(unbiased){
            B[thread_id_x] = sum/((float)size_reduction-1.0f);
        } else {
            B[thread_id_x] = sum/(float)size_reduction;
        }
    }
}

__global__ void gpu_norm_fro(float *A, float *B, int *map, int size, int size_reduction){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp = 0.0f;
        float val;
        for(int i=0; i<size_reduction; i++){
            val = A[map[thread_id_x*size_reduction+i]];
            tmp += val*val;
        }

        B[thread_id_x] = sqrt(tmp);
    }
}

__global__ void gpu_mode(float *A, float *B, int *map, int size, int size_reduction){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x<size) {
        float tmp = 1.0f;
        for(int i=0; i<size_reduction; i++){
            tmp *= A[map[thread_id_x*size_reduction+i]];
        }

        B[thread_id_x] = tmp;
    }
}


/* PREVIOUS REDUCES ***********************************/

__global__ void reduce_mean(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    atomicAdd(&(B[map[thread_id_x]]),A[thread_id_x]);
  }

}

__global__ void reduce_op_sum(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]+=B[map[thread_id_x]];
  }
}

__global__ void reduce_op_diff(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]-=B[map[thread_id_x]];
  }

}
__global__ void reduce_op_mult(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]*=B[map[thread_id_x]];
  }

}
__global__ void reduce_op_div(float *A,float *B,int *map,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size) {
    A[thread_id_x]/=B[map[thread_id_x]];
  }

}


//dim3 dimGrid(RD->index.size());
//dim3 dimBlock(1);
__global__ void reduction_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int rs)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;


  int j;
  float sum=0;
  float v,val;

  int i;

  int p=rs*blockIdx.x;


  for(j=0;j<rs;j++,p++) {
      v=I[ind[p]];
      if (m==2) {
          if (j==0) {val=v;i=p;}
          else if (v>val) {
              val=v;
              i=p;
          }
      }
      else if (m==3) {
        if (j==0) {val=v;i=p;}
        else if (v<val) {
            val=v;
            i=p;
        }
      }
      else sum+=v;
  }

  p=rs*blockIdx.x;
  // set in Output
  if (m<2) { // mean or sum
      if (m==0) sum/=d;
      if (keepdims) {
        for(j=0;j<rs;j++,p++)
            O[ind[p]]=sum;
      }
      else O[thread_id_x]=sum;
  }
  else { // rs or min
      if (keepdims) {
        for(j=0;j<rs;j++,p++) {
              O[ind[p]]=val;
              S[ind[p]]=i;
          }
      }
      else {
          O[thread_id_x]=val;
          S[thread_id_x]=i;
      }
  }

}


//dim3 dimGrid(RD->index.size());
//dim3 dimBlock(1);
__global__ void reduction_back_kernel(float *I,float *O,float *S,int m, int keepdims,int d,int *ind,int rs)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    int j;
    float val=0;
    int p;

    // set in Delta
    if (m>=2) {
      int p=S[thread_id_x];
      O[p]+=I[thread_id_x];
    }
    else {
      p=rs*blockIdx.x;
      if(keepdims) {
        for(j=0;j<rs;j++,p++)
          val+=I[ind[p]];
      }
      else val=I[thread_id_x];
      if (m==0) val/=d;

      p=rs*blockIdx.x;
      for(j=0;j<rs;j++,p++)
        O[ind[p]]+=val;
    }
}



////////////////////
// FOR SUM and MEAN
// Faster in Conv
///////////////////

//dim3 dimGrid(red_size);
//dim3 dimBlock(RD->index.size());

__global__ void reduction_permute(float *I,float *O,int *ind,int size)
{
  long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

  if (thread_id_x<size)
    O[thread_id_x]=I[ind[thread_id_x]];
}

__global__ void reduction_kernel_keep(float *red, float *O, int *ind, int size, int rsize)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    if (thread_id_x<size*rsize) {
        O[ind[thread_id_x]]=red[thread_id_x/rsize];
    }
}

__global__ void reduction_kernel_keep_inc(float *red, float *O, int *ind, int size, int rsize)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;
    if (thread_id_x<size*rsize) {
        O[ind[thread_id_x]]+=red[thread_id_x/rsize];
    }
}
