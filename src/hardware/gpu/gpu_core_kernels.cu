#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.2
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"


__global__ void fill_(float* a, float v, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        a[thread_id_x]=v;
    }
}

__global__ void fill(float *aptr,float *bptr,int t,int aini,int at,int bini,int bt,int tot,int inc){
    int i=blockIdx.x;
    int j=threadIdx.x;
    int k=blockIdx.y;

    int ap=(i*at)+((aini+j)*t)+k;
    int bp=(i*bt)+((bini+j)*t)+k;

    if (bp<tot){
        if (inc) {
            bptr[bp] += aptr[ap];
        } else { bptr[bp]=aptr[ap];}
    }

}


__global__ void mask(float* a, float v, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        a[thread_id_x]=a[thread_id_x]<v;
    }
}


__global__ void select(float* A, float* B, int batch, int* A_stride, int* B_stride, int* indices){
    long int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    long int size = batch * B_stride[0];  // B is the small

    if (thread_id_x < size){
        int b = thread_id_x / B_stride[0] % batch;

        int A_str_batch = b * A_stride[0];
        int B_str_batch = b * B_stride[0];

        int i = thread_id_x % B_stride[0];
        int A_pos = A_str_batch + indices[i];
        int B_pos = B_str_batch + i;

        B[B_pos] = A[A_pos];
    }
}

__global__ void select_back(float* A, float* B, int batch, int* A_stride, int* B_stride, int* indices){
    long int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    long int size = batch * A_stride[0];  // A is the small

    if (thread_id_x < size){
        int b = thread_id_x / A_stride[0] % batch;

        int A_str_batch = b * A_stride[0];
        int B_str_batch = b * B_stride[0];

        int i = thread_id_x % A_stride[0];
        int A_pos = A_str_batch + i;
        int B_pos = B_str_batch + indices[i];

        B[B_pos] += A[A_pos];
    }
}