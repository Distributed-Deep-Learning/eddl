#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.1
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"


__global__ void fill(float *aptr,float *bptr,int t,int aini,int at,int bini,int bt,int tot,int inc)
{
  int i=blockIdx.x;
  int j=threadIdx.x;
  int k=blockIdx.y;

  int ap=(i*at)+((aini+j)*t)+k;
  int bp=(i*bt)+((bini+j)*t)+k;

  if (bp<tot)
    if (inc) bptr[bp]+=aptr[ap];
    else bptr[bp]=aptr[ap];

}


__global__ void mask(float* a, float v, long int size)
{

 long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

 if (thread_id_x < size)
   a[thread_id_x]=a[thread_id_x]<v;

}


__global__ void set(float* a, float v, long int size)
{

    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size)
        a[thread_id_x]=v;

}
