#include "hip/hip_runtime.h"
/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.2
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/


#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"


__global__ void fill_(float* a, float v, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        a[thread_id_x]=v;
    }
}

__global__ void fill(float *aptr,float *bptr,int t,int aini,int at,int bini,int bt,int tot,int inc){
    int i=blockIdx.x;
    int j=threadIdx.x;
    int k=blockIdx.y;

    int ap=(i*at)+((aini+j)*t)+k;
    int bp=(i*bt)+((bini+j)*t)+k;

    if (bp<tot){
        if (inc) {
            bptr[bp] += aptr[ap];
        } else { bptr[bp]=aptr[ap];}
    }

}


__global__ void mask(float* a, float v, long int size){
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size){
        a[thread_id_x]=a[thread_id_x]<v;
    }

}


__global__ void select(float* A, float* B, int batch, int depth, int orows, int ocols, int* A_stride, int* B_stride, int* indices){
    long int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
    long int size = batch * depth*orows*ocols;

    if (thread_id_x < size){
        int b = thread_id_x / B_stride[0] % batch;

        int A_str_batch = b * A_stride[0];
        int B_str_batch = b * B_stride[0];

        int i = thread_id_x % B_stride[0];
        int A_pos = A_str_batch + indices[i];
        int B_pos = B_str_batch + i;

        B[thread_id_x] = A[A_pos];
    }
}

__global__ void select_back(float* A, float* B, int batch, int depth, int orows, int ocols, int* A_stride, int* B_stride, int* indices){
    int b = thread_id_x / B_stride[0] % batch;

    int A_str_batch = b * A_stride[0];
    int B_str_batch = b * B_stride[0];

    int i = thread_id_x % B_stride[0];
    int A_pos = A_str_batch + indices[i];
    int B_pos = B_str_batch + i;

    A[A_pos] += B[thread_id_x];
}