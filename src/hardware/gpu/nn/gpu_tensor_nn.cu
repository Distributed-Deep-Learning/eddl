/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.2
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>


#include "gpu_nn.h"
#include "gpu_nn_kernels.h"

#include "../gpu_hw.h"
#include "../gpu_tensor.h"
#include "../gpu_kernels.h"

#include "../../../tensor/tensor.h"
#include "../../../descriptors/descriptors.h"


void gpu_repeat_nn(Tensor *A, Tensor *B, vector<int> size){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(B);

    // Copy vector from host to device
    int *d_size; hipMalloc((int**)&d_size, 2*sizeof(int));
    hipMemcpy(d_size, size.data(), 2*sizeof(int), hipMemcpyHostToDevice);

    repeat_nn_k<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], A->shape[1], A->shape[2], A->shape[3], B->ptr, B->shape[2], B->shape[3], d_size);

    hipFree(d_size);
    check_cuda(hipDeviceSynchronize(), "repeat_nn_k");
}

void gpu_d_repeat_nn(Tensor *D, Tensor *A, vector<int> size){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(D);

    // Copy vector from host to device
    int *d_size; hipMalloc((int**)&d_size, 2*sizeof(int));
    hipMemcpy(d_size, size.data(), 2*sizeof(int), hipMemcpyHostToDevice);

    d_repeat_nn_k<<<dimGrid,dimBlock>>>(D->ptr, D->shape[0], D->shape[1], D->shape[2], D->shape[3], A->ptr, A->shape[2], A->shape[3], d_size);

    hipFree(d_size);
    check_cuda(hipDeviceSynchronize(), "d_repeat_nn_k");
}
