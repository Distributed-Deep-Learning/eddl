#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "../tensor_cuda.h"
#include "../tensor_kernels.h"
#include "../gpu_hw.h"

#include "../../../tensor/tensor.h"
#include "../../../descriptors/descriptors.h"


// MAX THREADS PER BLOCK
#define MAX_TPB 1024
#define setDims(A) int r,c;r=(A->size/MAX_TPB);if (r==0) {r=1;c=A->size;}else {if (A->size%MAX_TPB) r++;c=MAX_TPB;}dim3 dimGrid(r);dim3 dimBlock(c);

extern hipblasHandle_t hcublas[64];
extern hiprandGenerator_t random_generator[64];

void gpu_accuracy(Tensor *A,Tensor *B,int *acc){
  int device=A->gpu_device;
  hipSetDevice(device);
  int r,c;

  r=A->shape[0];
  c=A->size/r;

  dim3 dimGrid(r);
  dim3 dimBlock(c);

  float* max_row=gpu_create_tensor(device,r);

  int *a;
  check_cuda(hipMalloc((void**)&a,sizeof(int)),"error hipMalloc in accuracy");
  hipMemset(a, 0, sizeof(int));

  accuracy<<<dimBlock,dimGrid>>>(A->ptr,B->ptr,max_row,c,r,a);
  check_cuda(hipMemcpy(acc,a,sizeof(int),hipMemcpyDeviceToHost),"error copy in accuracy");

  hipFree(a);
  gpu_delete_tensor(device,max_row);

}

