/*
* EDDL Library - European Distributed Deep Learning Library.
* Version: 0.3
* copyright (c) 2019, Universidad Politécnica de Valencia (UPV), PRHLT Research Centre
* Date: October 2019
* Author: PRHLT Research Centre, UPV, (rparedes@prhlt.upv.es), (jon@prhlt.upv.es)
* All rights reserved
*/

#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "hardware/gpu/gpu_nn.h"
#include "hardware/gpu/gpu_nn_kernels.h"

#include "hardware/gpu/gpu_hw.h"
#include "hardware/gpu/gpu_tensor.h"
#include "hardware/gpu/gpu_kernels.h"

#include "tensor/tensor.h"
#include "descriptors/descriptors.h"


void gpu_cent(Tensor *A,Tensor *B,Tensor *C){

  int device=A->gpu_device;
  hipSetDevice(device);
  setDims(A);

  cent<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->size);
  check_cuda(hipDeviceSynchronize(),"gpu_cent");
}
