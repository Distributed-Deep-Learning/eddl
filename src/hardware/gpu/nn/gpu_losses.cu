#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_nn.h"
#include "../gpu_hw.h"
#include "../gpu_tensor.h"
#include "../gpu_kernels.h"

#include "../../../tensor/tensor.h"
#include "../../../descriptors/descriptors.h"


void gpu_cent(Tensor *A,Tensor *B,Tensor *C){

  int device=A->gpu_device;
  hipSetDevice(device);
  setDims(A);

  cent<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->size);
  check_cuda(hipDeviceSynchronize(),"gpu_cent");
}
