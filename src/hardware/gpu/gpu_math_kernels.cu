#include "hip/hip_runtime.h"

/////////////////////////////////////////////////////////////////////////////
// This file is part of EDDLL an European Distributed Deep Learning Library.
// Developed within the DeepHealth project.
// Boosting AI in Europe.
//
// Main authors and developers:
//      Roberto Paredes: rparedes@prhlt.upv.es
//      Joan Ander Gómez: jon@prhlt.upv.es
//
//
// Collaborators:
//      Juan Maroñas: jmaronas@prhlt.upv.es
//
//
// To collaborate please contact rparedes@prhlt.upv.es
//
/////////////////////////////////////////////////////////////////////////////

#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include "gpu_kernels.h"


__global__ void abs_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=fabsf(a[thread_id_x]);
}

__global__ void acos_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=acosf(a[thread_id_x]);
}

__global__ void add_(float* a, long int rows, long int cols, float v){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]+=v;
}

__global__ void asin_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=asinf(a[thread_id_x]);
}

__global__ void atan_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=atanf(a[thread_id_x]);
}
__global__ void ceil_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=ceilf(a[thread_id_x]);
}

__global__ void clamp_(float* a, long int rows, long int cols, float min, float max){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        if (a[thread_id_x] < min){
            a[thread_id_x] = min;
        } else if(a[thread_id_x] > max){
            a[thread_id_x] = max;
        }
}

__global__ void cos_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=cosf(a[thread_id_x]);
}

__global__ void cosh_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=coshf(a[thread_id_x]);
}

__global__ void exp_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=expf(a[thread_id_x]);
}

__global__ void floor_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=floorf(a[thread_id_x]);
}

__global__ void log_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=logf(a[thread_id_x]);
}

__global__ void log2_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=log2f(a[thread_id_x]);
}

__global__ void log10_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=log10f(a[thread_id_x]);
}

__global__ void logn_(float* a, long int rows, long int cols, float n){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=logf(a[thread_id_x])/logf(n);
}

__global__ void mod_(float* a, long int rows, long int cols, float v){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=fmodf(a[thread_id_x], v);
}

__global__ void mult_(float* a, long int rows, long int cols, float v){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x] *= v;
}

__global__ void normalize_(float* a, long int rows, long int cols, float min_ori, float max_ori, float min, float max){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=(max-min)/(max_ori-min_ori) * (a[thread_id_x]-min_ori) + min;
}

__global__ void pow_(float* a, long int rows, long int cols, float exp){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=powf(a[thread_id_x], exp);
}

__global__ void reciprocal_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=1.0f/a[thread_id_x];
}

__global__ void remainder_(float* a, long int rows, long int cols, float v){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x] = (int)(a[thread_id_x]/v);
}

__global__ void round_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=roundf(a[thread_id_x]);
}

__global__ void rsqrt_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=1.0f/sqrtf(a[thread_id_x]);
}

__global__ void sigmoid_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x] = expf(a[thread_id_x])/(expf(a[thread_id_x])+1.0f);
}

__global__ void sign_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops){
        if(a[thread_id_x] > 0.0f){
            a[thread_id_x] = 1.0f;
        }else if(a[thread_id_x] < 0.0f){
            a[thread_id_x] = -1.0f;
        }else{
            a[thread_id_x] = 0.0f;
        }
    }
}

__global__ void sin_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=sinf(a[thread_id_x]);
}

__global__ void sinh_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=sinhf(a[thread_id_x]);
}

__global__ void sqr_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]*=a[thread_id_x];
}

__global__ void sqrt_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=sqrtf(a[thread_id_x]);
}

__global__ void tan_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=tanf(a[thread_id_x]);
}

__global__ void tanh_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]=tanhf(a[thread_id_x]);
}

__global__ void trunc_(float* a, long int rows, long int cols){
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        a[thread_id_x]= (int)(a[thread_id_x]);
}



///////////////////////////////////////////

__global__ void reduce_array_sum(float* a, long int ops, float* result)
{
  long int thread_id_x = threadIdx.x+(blockDim.x*blockIdx.x);

  if (thread_id_x < ops){
    atomicAdd(result,a[thread_id_x]);
  }
}

///////////////////////////////////////////

__global__ void addc(float scA,float* a,float scB,float *b, float *c,long int incC, long int size)
{
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < size) {
        if (incC) c[thread_id_x]+=scA*a[thread_id_x]+scB*b[thread_id_x];
        else c[thread_id_x]=scA*a[thread_id_x]+scB*b[thread_id_x];
    }
}

__global__ void el_mult(float* a, float *b, float *c, long int incC, long int rows, long int cols)
{
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        if (incC) c[thread_id_x]+=a[thread_id_x]*b[thread_id_x];
        else c[thread_id_x]=a[thread_id_x]*b[thread_id_x];
}

__global__ void el_div(float* a, float *b, float *c, long int incC, long int rows, long int cols)
{
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        if (incC) c[thread_id_x]+=a[thread_id_x]/(b[thread_id_x]);
        else c[thread_id_x]=a[thread_id_x]/(b[thread_id_x]);
}


__global__ void sum_mat_row(float* a, float* b, float* c, long int rows, long int cols)
{
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        c[thread_id_x]=a[thread_id_x]+b[thread_id_x%cols];

}

__global__ void sum_mat_col(float* a, float* b, float* c, long int rows, long int cols)
{
    long int ops=rows*cols;
    long int thread_id_x = threadIdx.x+blockIdx.x*blockDim.x;

    if (thread_id_x < ops)
        c[thread_id_x]=a[thread_id_x]+b[thread_id_x/cols];

}


__global__ void reduce_sum2D(float *a,float *b,long int rows,long int cols,long int axis)
{
  long int ops=rows*cols;
  long int thread_id_x = threadIdx.x+(blockDim.x*blockIdx.x);

  if (thread_id_x < ops){
    if (axis==0)
      atomicAdd(&(b[thread_id_x%cols]),a[thread_id_x]);
    else
      atomicAdd(&(b[thread_id_x/cols]),a[thread_id_x]);
  }

}
