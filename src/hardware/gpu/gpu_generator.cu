#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


void gpu_rand_uniform(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

  gpu_mult(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

}


void gpu_rand_signed_uniform(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_signed_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_signed_uniform");

  gpu_mult(A,2*v);
  gpu_add(A,-v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_signed_uniform");

}


void gpu_rand_normal(Tensor *A, float m, float s){
  int device=A->gpu_device;
  hipSetDevice(device);

  if (A->size%2) {
    gpu_set(A,0.0);
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->size-1,m,s),"gpu_rand_normal");
  }
  else
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->size,m,s),"gpu_rand_normal");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_normal");

}


void gpu_rand_binary(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_binary");

  gpu_mask(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_binary");

}

