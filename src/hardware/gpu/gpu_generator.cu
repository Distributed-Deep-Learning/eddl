#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "tensor_cuda.h"
#include "tensor_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


// MAX THREADS PER BLOCK
#define MAX_TPB 1024
#define setDims(A) int r,c;r=(A->size/MAX_TPB);if (r==0) {r=1;c=A->size;}else {if (A->size%MAX_TPB) r++;c=MAX_TPB;}dim3 dimGrid(r);dim3 dimBlock(c);

extern hipblasHandle_t hcublas[64];
extern hiprandGenerator_t random_generator[64];

void gpu_rand_uniform(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

  gpu_mult(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_uniform");

}


void gpu_rand_signed_uniform(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_signed_uniform");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_signed_uniform");

  gpu_mult(A,2*v);
  gpu_add(A,-v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_signed_uniform");

}


void gpu_rand_normal(Tensor *A, float m, float s){
  int device=A->gpu_device;
  hipSetDevice(device);

  if (A->size%2) {
    gpu_set(A,0.0);
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->size-1,m,s),"gpu_rand_normal");
  }
  else
    check_curand(hiprandGenerateNormal(random_generator[device],A->ptr,A->size,m,s),"gpu_rand_normal");

  check_cuda(hipDeviceSynchronize(),"gpu_rand_normal");

}


void gpu_rand_binary(Tensor *A, float v){
  int device=A->gpu_device;
  hipSetDevice(device);

  check_curand(hiprandGenerateUniform(random_generator[device],A->ptr,A->size),"gpu_rand_binary");

  gpu_mask(A,v);

  check_cuda(hipDeviceSynchronize(),"gpu_rand_binary");

}

