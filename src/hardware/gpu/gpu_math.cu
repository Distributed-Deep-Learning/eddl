#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include "gpu_tensor.h"
#include "gpu_kernels.h"
#include "gpu_hw.h"

#include "../../tensor/tensor.h"
#include "../../descriptors/descriptors.h"


// CPU: Math (in-place) ********************************************
void gpu_abs_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    abs_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "abs_");
}

void gpu_acos_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    acos_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "acos_");
}

void gpu_add_(Tensor *A, float v) {
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  add_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, v);
  check_cuda(hipDeviceSynchronize(), "add_");
}

void gpu_asin_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    asin_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "asin_");
}

void gpu_atan_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    atan_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "atan_");
}

void gpu_ceil_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    ceil_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "ceil_");
}

void gpu_clamp_(Tensor *A, float min, float max){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    clamp_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, min, max);
    check_cuda(hipDeviceSynchronize(), "clamp_");
}

void gpu_cos_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    cos_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "cos_");
}

void gpu_cosh_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    cosh_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "cosh_");
}

void gpu_exp_(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  exp_<<<dimGrid,dimBlock>>>(A->ptr,A->shape[0],c);
  check_cuda(hipDeviceSynchronize(),"exp_");

}

void gpu_floor_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    floor_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "floor_");
}


void gpu_log_(Tensor *A) {
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  log_<<<dimGrid,dimBlock>>>(A->ptr,A->shape[0], c);
  check_cuda(hipDeviceSynchronize(), "log_");

}


void gpu_log2_(Tensor *A) {
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    log2_<<<dimGrid,dimBlock>>>(A->ptr,A->shape[0], c);
    check_cuda(hipDeviceSynchronize(),"log2_");
}


void gpu_log10_(Tensor *A) {
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    log10_<<<dimGrid,dimBlock>>>(A->ptr,A->shape[0], c);
    check_cuda(hipDeviceSynchronize(),"log10_");
}


void gpu_logn_(Tensor *A, float n){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    logn_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, n);
    check_cuda(hipDeviceSynchronize(), "logn_");
};

void gpu_mod_(Tensor *A, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    mod_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, v);
    check_cuda(hipDeviceSynchronize(), "mod_");
}

void gpu_mult_(Tensor *A, float v) {
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  mult_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, v);
  check_cuda(hipDeviceSynchronize(),"mult_");

}

void gpu_normalize_(Tensor *A, float min, float max){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    // TODO: Temp
    float min_ori = 0;
    float max_ori = 10000;

    normalize_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, min_ori, max_ori, min, max);
    check_cuda(hipDeviceSynchronize(), "normalize_");
}

void gpu_pow_(Tensor *A, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    pow_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, v);
    check_cuda(hipDeviceSynchronize(), "pow_");
}


void gpu_reciprocal_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    reciprocal_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "reciprocal_");
}

void gpu_remainder_(Tensor *A, float v){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    remainder_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c, v);
    check_cuda(hipDeviceSynchronize(), "remainder_");
}

void gpu_round_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    round_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "round_");
}

void gpu_rsqrt_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    rsqrt_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "rsqrt_");
}

void gpu_sigmoid_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    sigmoid_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "sigmoid_");
}

void gpu_sign_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    sign_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "sign_");
}

void gpu_sin_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    sin_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "sin_");
}

void gpu_sinh_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    sinh_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "sinh_");
}


void gpu_sqr_(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  sqr_<<<dimGrid,dimBlock>>>(A->ptr,A->shape[0],c);
  check_cuda(hipDeviceSynchronize(),"sqr_");

}

void gpu_sqrt_(Tensor *A) {

  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  sqrt_<<<dimGrid,dimBlock>>>(A->ptr,A->shape[0],c);
  check_cuda(hipDeviceSynchronize(),"sqrt_");
}

void gpu_tan_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    tan_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "tan_");
}

void gpu_tanh_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    tanh_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "tanh_");
}

void gpu_trunc_(Tensor *A){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);

    trunc_<<<dimGrid,dimBlock>>>(A->ptr, A->shape[0], c);
    check_cuda(hipDeviceSynchronize(), "trunc_");
}

// CPU: Math (static) ********************************************


void gpu_addc(float scA,Tensor *A, float scB,Tensor *B, Tensor *C,int incC){
    int device=A->gpu_device;
    hipSetDevice(device);

    setDims(A);


    addc<<<dimGrid,dimBlock>>>(scA,A->ptr,scB,B->ptr,C->ptr,incC,A->size);
    check_cuda(hipDeviceSynchronize(),"addc");
}


void gpu_mult2D(Tensor *A, int tA, Tensor *B, int tB, Tensor *C,int incC){
  int device=A->gpu_device;
  hipSetDevice(device);

  float alfa=1.0;
  float beta=(float)incC;

  hipblasOperation_t trA = HIPBLAS_OP_N;
  hipblasOperation_t trB = HIPBLAS_OP_N;

  int ldA=A->shape[1];
  int ldB=B->shape[1];
  int ldC=B->shape[1];
  int m=B->shape[1];
  int n=A->shape[0];
  int k=B->shape[0];


  if (tA)
  {
    trA = HIPBLAS_OP_T;
  	n=A->shape[1];
  }
  if (tB)
    {
  	trB = HIPBLAS_OP_T;
    m=B->shape[0];
  	k=B->shape[1];
    ldC=B->shape[0];
    }

  check_cublas(hipblasSgemm(hcublas[device],trB,trA,m,n,k,&alfa,B->ptr,ldB,A->ptr,ldA,&beta,C->ptr,ldC),"mult2D");

}


void gpu_el_div(Tensor *A, Tensor *B, Tensor *C,int incC) {
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  el_mult<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,incC,A->shape[0],r);

  check_cuda(hipDeviceSynchronize(),"gpu_el_div");
}


void gpu_el_mult(Tensor *A, Tensor *B, Tensor *C,int incC){
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  el_mult<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,incC,A->shape[0],c);

  check_cuda(hipDeviceSynchronize(),"gpu_el_mult");
}


void gpu_sum2D_rowwise(Tensor *A, Tensor *B, Tensor *C){
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);


  sum_mat_row<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->shape[0],A->shape[1]);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");

}


void gpu_sum2D_colwise(Tensor *A, Tensor *B, Tensor *C){
  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  sum_mat_col<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,C->ptr,A->shape[0],A->shape[1]);

  check_cuda(hipDeviceSynchronize(),"sum2D_rowwise");

}

// CPU: Should be reductions ***************************************

void gpu_total_sum(Tensor *A, float *tot)
{
  float *total;
  int device=A->gpu_device;
  hipSetDevice(device);
  float t=0;


  setDims(A);

  check_cuda(hipMalloc((void**)&total,sizeof(float)),"create float in sum");

  check_cuda(hipMemcpy(total,&t,sizeof(float),hipMemcpyHostToDevice),"error copy in sum");

  reduce_array_sum<<<dimGrid,dimBlock>>>(A->ptr,A->size,total);

  check_cuda(hipMemcpy(tot,total,sizeof(float),hipMemcpyDeviceToHost),"error copy in sum");

  check_cuda(hipFree(total),"delete float in sum");
}




void gpu_sum2D(float scA,Tensor *A, float scB,Tensor *B, Tensor *C,int incC){
    int device=A->gpu_device;
    hipSetDevice(device);

    int m=A->shape[1];
    int n=B->shape[0];
    int ldA=A->shape[1];
    int ldB=B->shape[1];
    int ldC=A->shape[1];

    float alfa=scA;
    float beta=scB;
    float one=1.0;


    if (incC){
        check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,A->ptr,ldA,&one,C->ptr,ldB,C->ptr,ldC),"sum2D");
        check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,B->ptr,ldA,&one,C->ptr,ldB,C->ptr,ldC),"sum2D");
    }
    else
        check_cublas(hipblasSgeam(hcublas[device],HIPBLAS_OP_N,HIPBLAS_OP_N, m,n,&alfa,A->ptr,ldA,&beta,B->ptr,ldB,C->ptr,ldC),"sum2D");

}


void gpu_reduce_sum2D(Tensor *A,Tensor *B,int axis,int incB){

  int device=A->gpu_device;
  hipSetDevice(device);

  setDims(A);

  if (!incB) gpu_set(B,0.0);

  reduce_sum2D<<<dimGrid,dimBlock>>>(A->ptr,B->ptr,A->shape[0],A->shape[1],axis);


  check_cuda(hipDeviceSynchronize(),"reduce_sum2D");
}
